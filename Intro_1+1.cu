#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add( int a, int b, int* c )
{
    *c = a + b;

    return;
}

int main(int argc, char** argv)
{
    int c;
    int* dev_c;

    hipMalloc((void**)&dev_c, sizeof(int) );

    add<<<1,380>>>(1, 2, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("a + b = %d\n", c);

    hipFree(dev_c);

    return 0;
}
